#include "hip/hip_runtime.h"
#include "gaussian.cuh"

__constant__ unsigned char dev_const_conv_kernel[3][3];

__global__ void k_1D_gf(unsigned char* input, int rows, int cols, int mask_dim)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int new_val = 0;
	int offset = 1;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += conv_kernel[i][j] * input[(tx - offset + i) * cols + ty - offset + j];
			}
		}
		input[threadId] = static_cast<uchar>(new_val / 16);
	}
}

__global__ void k_1D_gf_load_balance(unsigned char* input, int rows, int cols, int load)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * load;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = (tx * cols + ty);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int offset = 1;

	for (int i = 0; i < load; i++) {
		int new_val = 0;
		int _tx = tx;
		int _ty = ty + i;

		for (int r = 0; r < 3; r++)
		{
			for (int c = 0; c < 3; c++)
			{
				if ((_tx > 0 && _tx < rows - 1) && (_ty > 0 && _ty < cols - 1))
				{
					new_val += conv_kernel[r][c] * input[(_tx - offset + r) * cols + (_ty - offset + c)];
				}
				else
				{
					return;
				}
			}
		}
		input[(_tx * cols + _ty)] = static_cast<uchar>(new_val / 16);
	}
}

__global__ void k_1D_gf_vectorized(unsigned char* input, int rows, int cols, int load)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * load;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = (tx * cols + ty);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int offset = 1;
	int vals[4] = { 0 };

	for (int i = 0; i < load; i++) {
		int new_val = 0;
		int _tx = tx;
		int _ty = ty + i;

		for (int r = 0; r < 3; r++)
		{
			for (int c = 0; c < 3; c++)
			{
				if ((_tx > 0 && _tx < rows - 1) && (_ty > 0 && _ty < cols - 1))
				{
					vals[i] += conv_kernel[r][c] * input[(_tx - offset + r) * cols + (_ty - offset + c)];
				}
				else
				{
					return;
				}
			}
		}
	}
	switch (load)
	{
	case 2:
		reinterpret_cast<uchar2*>(&input[(tx * cols + ty)])[0] = make_uchar2(vals[0] / 16, vals[1] / 16);
		break;
	case 3:
		reinterpret_cast<uchar3*>(&input[(tx * cols + ty)])[0] = make_uchar3(vals[0] / 16, vals[1] / 16, vals[2] / 16);
		break;
	case 4:
		reinterpret_cast<uchar4*>(&input[(tx * cols + ty)])[0] = make_uchar4(vals[0] / 16, vals[1] / 16, vals[2] / 16, vals[3] / 16);
		break;
	default:
		break;
	}
}

__global__ void k_1D_gf_unroll(unsigned char* input, int rows, int cols, int mask_dim)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int new_val = 0;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		new_val += conv_kernel[0][0] * input[(tx - 1) * cols + ty - 1];
		new_val += conv_kernel[0][1] * input[(tx - 1) * cols + ty];
		new_val += conv_kernel[0][2] * input[(tx - 1) * cols + ty + 1];
		new_val += conv_kernel[1][0] * input[tx * cols + ty - 1];
		new_val += conv_kernel[1][1] * input[tx * cols + ty];
		new_val += conv_kernel[1][2] * input[tx * cols + ty + 1];
		new_val += conv_kernel[2][0] * input[(tx + 1) * cols + ty - 1];
		new_val += conv_kernel[2][1] * input[(tx + 1) * cols + ty];
		new_val += conv_kernel[2][2] * input[(tx + 1) * cols + ty + 1];

		input[threadId] = static_cast<uchar>(new_val / 16);
	}
}

__global__ void k_1D_gf_prefetch(unsigned char* input, int rows, int cols, int mask_dim)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int new_val = 0;
	int offset = 1;

	int x_index = tx - offset;
	int y_index = ty - offset;
	unsigned char pixel = input[x_index * cols + y_index];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) { /*conv element prefetch*/
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += conv_kernel[i][j] * pixel;
				pixel = input[x_index * cols + (++y_index)];
			}
			y_index = ty - offset;
			pixel = input[(++x_index) * cols + y_index];
		}
		input[threadId] = static_cast<uchar>(new_val / 16);
	}
}

__global__ void k_1D_gf_constant(unsigned char* input, int rows, int cols, int mask_dim)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	int new_val = 0;
	int offset = 1;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += dev_const_conv_kernel[i][j] * input[(tx - offset + i) * cols + ty - offset + j];
			}
		}
		input[threadId] = static_cast<uchar>(new_val / 16);
	}
}

__global__ void k_1D_gf_shared(unsigned char* input, int rows, int cols, int mask_dim)
{
	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	__shared__  unsigned char cache[34][36];

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	unsigned int bx = threadIdx.y;
	unsigned int by = threadIdx.x;

	unsigned int cy = by + 1;
	unsigned int cx = bx + 1;

	cache[cx][cy] = input[tx * cols + ty];

	if (cx == 1) {
		cache[0][cy] = input[((tx - 1) * cols + ty)];
	}
	if (cx == 32) {
		cache[33][cy] = input[((tx + 1) * cols + ty)];
	}
	if (cy == 1) {
		cache[cx][0] = input[((tx)*cols + ty - 1)];
	}
	if (cy == 32) {
		cache[cx][33] = input[((tx)*cols + ty + 1)];
	}
	__syncthreads();

	int new_val = 0;
	int offset = 1;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += conv_kernel[i][j] * cache[(cx - offset + i)][cy - offset + j];
			}
		}
	}
	else {
		return;
	}

	input[threadId] = static_cast<uchar>(new_val / 16);
}

__global__ void k_1D_gf_combined(unsigned char* input, int rows, int cols, int mask_dim)
{
	__shared__  unsigned char cache[34][36];

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	unsigned int bx = threadIdx.y;
	unsigned int by = threadIdx.x;

	unsigned int cy = by + 1;
	unsigned int cx = bx + 1;

	cache[cx][cy] = input[tx * cols + ty];

	if (cx == 1) {
		cache[0][cy] = input[((tx - 1) * cols + ty)];
	}
	if (cx == 32) {
		cache[33][cy] = input[((tx + 1) * cols + ty)];
	}
	if (cy == 1) {
		cache[cx][0] = input[((tx)*cols + ty - 1)];
	}
	if (cy == 32) {
		cache[cx][33] = input[((tx)*cols + ty + 1)];
	}

	__syncthreads();
	int new_val = 0;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		new_val += dev_const_conv_kernel[0][0] * cache[cx - 1][cy - 1];
		new_val += dev_const_conv_kernel[0][1] * cache[cx - 1][cy];
		new_val += dev_const_conv_kernel[0][2] * cache[cx - 1][cy + 1];

		new_val += dev_const_conv_kernel[1][0] * cache[cx][cy - 1];
		new_val += dev_const_conv_kernel[1][1] * cache[cx][cy];
		new_val += dev_const_conv_kernel[1][2] * cache[cx][cy + 1];

		new_val += dev_const_conv_kernel[2][0] * cache[cx + 1][cy - 1];
		new_val += dev_const_conv_kernel[2][1] * cache[cx + 1][cy];
		new_val += dev_const_conv_kernel[2][2] * cache[cx + 1][cy + 1];
	}
	else {
		return;
	}

	input[threadId] = static_cast<uchar>(new_val / 16);
}

float gf_1d_gpu(cv::Mat* output_img, GAUSSIAN ver)
{
	unsigned char* gpu_input = nullptr;
	unsigned char* output = output_img->data;

	unsigned int cols = (*output_img).cols;
	unsigned int rows = (*output_img).rows;
	unsigned int size = cols * rows * sizeof(unsigned char);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	const int mask_dim = 3;

	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	hipHostRegister(output, size, 0);

	CHECK_CUDA_ERROR(hipMalloc((unsigned char**)&gpu_input, size));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_input, output, size, hipMemcpyHostToDevice));

	switch (ver)
	{
	default:
		break;
	case GAUSSIAN_default:
		k_1D_gf << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_unroll:
		k_1D_gf_unroll << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_prefetch:
		k_1D_gf_prefetch << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_const_conv_kernel), conv_kernel, sizeof(uchar) * 3 * 3));
		k_1D_gf_constant << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_shared:
		k_1D_gf_shared << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_combined:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_const_conv_kernel), conv_kernel, sizeof(uchar) * 3 * 3));
		k_1D_gf_combined << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_load_balance:
		{
			int load = 3;
			dim3 grid_load_balance(((cols / load) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
			k_1D_gf_load_balance << <grid_load_balance, block >> > (gpu_input, rows, cols, load);
		}
		break;
	case GAUSSIAN_vectorized:
		{
			int load = 3;
			dim3 grid_load_balance(((cols / load) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
			k_1D_gf_vectorized << <grid_load_balance, block >> > (gpu_input, rows, cols, load);
		}
		break;
	}
	CHECK_CUDA_ERROR(hipMemcpy(output, gpu_input, size, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);

	hipHostUnregister(output);
	hipFree(gpu_input);
	hipDeviceReset();
	return elapsed;
}

__global__ void k_3D_gf(unsigned char* input, int rows, int cols, int mask_dim)
{
	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	int new_val = 0;
	int offset_x = 1, offset_y = 3;

	if ((tx > 2 && tx < rows - 2) && (ty > 2 && ty < cols - 2)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += conv_kernel[i][j] * input[(tx + i - offset_x) * cols + (ty + (j * 3) - offset_y)];
			}
		}
	}
	else {
		return;
	}

	input[threadId] = new_val >> 4;
}

__global__ void k_3D_gf_load_balance(unsigned char* input, int rows, int cols, int load)
{
	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * load;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	int offset_x = 1, offset_y = 3;

	for (int l = 0; l < load; l++) {
		int new_val = 0;
		int _tx = tx;
		int _ty = ty + l;

		if ((tx > 2 && tx < rows - 2) && (ty > 2 && ty < cols - 2)) {
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					new_val += conv_kernel[i][j] * input[(_tx + i - offset_x) * cols + (_ty + (j * 3) - offset_y)];
				}
			}
		}
		else {
			return;
		}
		input[_tx * cols + _ty] = new_val >> 4;
	}
}

__global__ void k_1D_gf_vectorized(unsigned char* input, int rows, int cols, int load)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * load;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = (tx * cols + ty);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int offset = 1;
	int vals[4] = { 0 };

	for (int i = 0; i < load; i++) {
		int new_val = 0;
		int _tx = tx;
		int _ty = ty + i;

		for (int r = 0; r < 3; r++)
		{
			for (int c = 0; c < 3; c++)
			{
				if ((_tx > 0 && _tx < rows - 1) && (_ty > 0 && _ty < cols - 1))
				{
					vals[i] += conv_kernel[r][c] * input[(_tx - offset + r) * cols + (_ty - offset + c)];
				}
				else
				{
					return;
				}
			}
		}
	}
	switch (load)
	{
	case 2:
		reinterpret_cast<uchar2*>(&input[(tx * cols + ty)])[0] = make_uchar2(vals[0] / 16, vals[1] / 16);
		break;
	case 3:
		reinterpret_cast<uchar3*>(&input[(tx * cols + ty)])[0] = make_uchar3(vals[0] / 16, vals[1] / 16, vals[2] / 16);
		break;
	case 4:
		reinterpret_cast<uchar4*>(&input[(tx * cols + ty)])[0] = make_uchar4(vals[0] / 16, vals[1] / 16, vals[2] / 16, vals[3] / 16);
		break;
	default:
		break;
	}
}


__global__ void k_3D_gf_vectorized(unsigned char* input, int rows, int cols, int load)
{
	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * load;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	int offset_x = 1, offset_y = 3;

	int vals[4] = {0};

	for (int l = 0; l < load; l++) {
		int new_val = 0;
		int _tx = tx;
		int _ty = ty + l;

		if ((tx > 2 && tx < rows - 2) && (ty > 2 && ty < cols - 2)) {
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					vals[i] += conv_kernel[i][j] * input[(_tx + i - offset_x) * cols + (_ty + (j * 3) - offset_y)];
				}
			}
		}
		else {
			return;
		}
		input[_tx * cols + _ty] = new_val >> 4;
	}
}


__global__ void k_3D_gf_unroll(unsigned char* input, int rows, int cols, int mask_dim)
{
	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	int new_val = 0;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		new_val += conv_kernel[0][0] * input[(tx - 1) * cols + ty - 3];
		new_val += conv_kernel[0][1] * input[(tx - 1) * cols + ty];
		new_val += conv_kernel[0][2] * input[(tx - 1) * cols + ty + 3];
		new_val += conv_kernel[1][0] * input[tx * cols + ty - 3];
		new_val += conv_kernel[1][1] * input[tx * cols + ty];
		new_val += conv_kernel[1][2] * input[tx * cols + ty + 3];
		new_val += conv_kernel[2][0] * input[(tx + 1) * cols + ty - 3];
		new_val += conv_kernel[2][1] * input[(tx + 1) * cols + ty];
		new_val += conv_kernel[2][2] * input[(tx + 1) * cols + ty + 3];
	}
	else {
		return;
	}

	input[threadId] = static_cast<uchar>(new_val / 16);
}

__global__ void k_3D_gf_prefetch(unsigned char* input, int rows, int cols, int mask_dim)
{
	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	int new_val = 0;
	int offset_x = 1, offset_y = 3;

	int x_index = tx - offset_x;
	int y_index = ty - offset_y;
	unsigned char pixel = input[x_index * cols + y_index];

	if ((tx > 2 && tx < rows - 2) && (ty > 2 && ty < cols - 2)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += conv_kernel[i][j] * pixel;
				y_index += 3;
				pixel = input[x_index * cols + y_index];
			}
			y_index = ty - offset_y;
			pixel = input[(++x_index) * cols + y_index];
		}
	}
	else {
		return;
	}
	input[threadId] = static_cast<uchar>(new_val / 16);
}


__global__ void k_3D_gf_constant(unsigned char* input, int rows, int cols, int mask_dim)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	int new_val = 0;
	int offset_x = 1, offset_y = 3;

	if ((tx > 2 && tx < rows - 2) && (ty > 2 && ty < cols - 2)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += dev_const_conv_kernel[i][j] * input[(tx + i - offset_x) * cols + (ty + (j * 3) - offset_y)];
			}
		}
	}
	else {
		return;
	}

	//if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
	//	new_val += dev_const_conv_kernel[0][0] * input[(tx - 1) * cols + ty - 3];
	//	new_val += dev_const_conv_kernel[0][1] * input[(tx - 1) * cols + ty];
	//	new_val += dev_const_conv_kernel[0][2] * input[(tx - 1) * cols + ty + 3];
	//	new_val += dev_const_conv_kernel[1][0] * input[tx * cols + ty - 3];
	//	new_val += dev_const_conv_kernel[1][1] * input[tx * cols + ty];
	//	new_val += dev_const_conv_kernel[1][2] * input[tx * cols + ty + 3];
	//	new_val += dev_const_conv_kernel[2][0] * input[(tx + 1) * cols + ty - 3];
	//	new_val += dev_const_conv_kernel[2][1] * input[(tx + 1) * cols + ty];
	//	new_val += dev_const_conv_kernel[2][2] * input[(tx + 1) * cols + ty + 3];
	//}
	//else {
	//	return;
	//}
	input[threadId] = static_cast<uchar>(new_val / 16);
}

__global__ void k_3D_gf_shared(unsigned char* input, int rows, int cols, int mask_dim)
{
	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	__shared__ unsigned char cache[34][38];

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	unsigned int bx = threadIdx.y;
	unsigned int by = threadIdx.x;

	unsigned int cy = by + 3;
	unsigned int cx = bx + 1;

	cache[cx][cy] = input[threadId];

	if (cx == 1) {
		cache[0][cy] = input[((tx - 1) * cols + ty)];
	}
	if (cx == 32) {
		cache[33][cy] = input[((tx + 1) * cols + ty)];
	}
	if (cy == 3) {
		cache[cx][0] = input[(tx * cols + ty - 3)];
		cache[cx][1] = input[(tx * cols + ty - 2)];
		cache[cx][2] = input[(tx * cols + ty - 1)];
	}
	if (cy == 34) {
		cache[cx][35] = input[(tx * cols + ty + 1)];
		cache[cx][36] = input[(tx * cols + ty + 2)];
		cache[cx][37] = input[(tx * cols + ty + 3)];
	}

	__syncthreads();

	int new_val = 0;
	int offset_x = 1, offset_y = 3;

	if ((tx > 2 && tx < rows - 2) && (ty > 2 && ty < cols - 2)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += conv_kernel[i][j] * cache[cx + i - offset_x][cy + (j * 3) - offset_y];
			}
		}
	}
	else {
		return;
	}

	//if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
	//	new_val += conv_kernel[0][0] * cache[cx - 1][cy - 3];
	//	new_val += conv_kernel[0][1] * cache[cx - 1][cy];
	//	new_val += conv_kernel[0][2] * cache[cx - 1][cy + 3];

	//	new_val += conv_kernel[1][0] * cache[cx][cy - 3];
	//	new_val += conv_kernel[1][1] * cache[cx][cy];
	//	new_val += conv_kernel[1][2] * cache[cx][cy + 3];

	//	new_val += conv_kernel[2][0] * cache[cx + 1][cy - 3];
	//	new_val += conv_kernel[2][1] * cache[cx + 1][cy];
	//	new_val += conv_kernel[2][2] * cache[cx + 1][cy + 3];
	//}
	//else {
	//	return;
	//}

	input[threadId] = static_cast<uchar>(new_val / 16);
}

__global__ void k_3D_gf_combined(unsigned char* input, int rows, int cols, int mask_dim)
{
	__shared__ unsigned char cache[34][38];

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty);

	unsigned int bx = threadIdx.y;
	unsigned int by = threadIdx.x;

	unsigned int cy = by + 3;
	unsigned int cx = bx + 1;

	cache[cx][cy] = input[threadId];

	if (cx == 1) {
		cache[0][cy] = input[((tx - 1) * cols + ty)];
	}
	if (cx == 32) {
		cache[33][cy] = input[((tx + 1) * cols + ty)];
	}
	if (cy == 3) {
		cache[cx][0] = input[(tx * cols + ty - 3)];
		cache[cx][1] = input[(tx * cols + ty - 2)];
		cache[cx][2] = input[(tx * cols + ty - 1)];
	}
	if (cy == 34) {
		cache[cx][35] = input[(tx * cols + ty + 1)];
		cache[cx][36] = input[(tx * cols + ty + 2)];
		cache[cx][37] = input[(tx * cols + ty + 3)];
	}

	__syncthreads();
	int new_val = 0;
	int offset_x = 1, offset_y = 3;

	if ((tx > 2 && tx < rows - 2) && (ty > 2 && ty < cols - 2)) {
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++) {
				new_val += dev_const_conv_kernel[i][j] * cache[cx + i - offset_x][cy + (j * 3) - offset_y];
			}
		}
	}
	else {
		return;
	}

	//if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
	//	new_val += dev_const_conv_kernel[0][0] * cache[cx - 1][cy - 3];
	//	new_val += dev_const_conv_kernel[0][1] * cache[cx - 1][cy];
	//	new_val += dev_const_conv_kernel[0][2] * cache[cx - 1][cy + 3];

	//	new_val += dev_const_conv_kernel[1][0] * cache[cx][cy - 3];
	//	new_val += dev_const_conv_kernel[1][1] * cache[cx][cy];
	//	new_val += dev_const_conv_kernel[1][2] * cache[cx][cy + 3];

	//	new_val += dev_const_conv_kernel[2][0] * cache[cx + 1][cy - 3];
	//	new_val += dev_const_conv_kernel[2][1] * cache[cx + 1][cy];
	//	new_val += dev_const_conv_kernel[2][2] * cache[cx + 1][cy + 3];
	//}
	//else {
	//	return;
	//}

	input[threadId] = static_cast<uchar>(new_val / 16);
}

float gf_3d_gpu(cv::Mat* output_img, GAUSSIAN ver)
{
	unsigned char* gpu_input = NULL;
	unsigned char* output = output_img->data;

	unsigned int cols = (*output_img).cols * 3;
	unsigned int rows = (*output_img).rows;
	unsigned int size = rows * cols * sizeof(unsigned char);

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	const uint mask_dim = 3;

	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	hipHostRegister(output, size, 0);

	CHECK_CUDA_ERROR(hipMalloc((unsigned char**)&gpu_input, size));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_input, output, size, hipMemcpyHostToDevice));

	switch (ver)
	{
	default:
		break;
	case GAUSSIAN_default:
		k_3D_gf << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_prefetch:
		k_3D_gf_prefetch << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_unroll:
		k_3D_gf_unroll << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_const_conv_kernel), conv_kernel, sizeof(uchar) * 3 * 3));
		k_3D_gf_constant << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_shared:
		k_3D_gf_shared << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_combined:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_const_conv_kernel), conv_kernel, sizeof(uchar) * 3 * 3));
		k_3D_gf_combined << <grid, block >> > (gpu_input, rows, cols, mask_dim);
		break;
	case GAUSSIAN_load_balance:
		{
			int load = 4;
			dim3 grid_load_balance(((cols / load) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
			k_3D_gf_load_balance << <grid_load_balance, block >> > (gpu_input, rows, cols, load);
		}
		break;
	case GAUSSIAN_vectorized:
		{
			int load = 4;
			dim3 grid_load_balance(((cols / load) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
			k_3D_gf_vectorized << <grid_load_balance, block >> > (gpu_input, rows, cols, load);
		}
	break;
	}

	CHECK_CUDA_ERROR(hipMemcpy(output, gpu_input, size, hipMemcpyDeviceToHost));
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);

	hipFree(gpu_input);
	hipDeviceReset();
	return elapsed;
}
