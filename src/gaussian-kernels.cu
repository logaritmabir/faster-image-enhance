#include "hip/hip_runtime.h"
#include "gaussian.cuh"

inline __device__ void shift_left(unsigned char arr[][3]) {
	arr[0][0] = arr[0][1];
	arr[1][0] = arr[1][1];
	arr[2][0] = arr[2][1];
	arr[0][1] = arr[0][2];
	arr[1][1] = arr[1][2];
	arr[2][1] = arr[2][2];
}

__constant__ unsigned char const_conv_kernel3x3[3][3];

__device__ unsigned char global_conv_kernel3x3[3][3] = {{1, 2, 1}, 
														{2, 4, 2}, 
														{1, 2, 1} };

__global__ void k_1D_gf_3x3_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		output[tx * cols + ty] = (global_conv_kernel3x3[0][0] * input[(tx - 1) * cols + ty - 1]
		 + global_conv_kernel3x3[0][1] * input[(tx - 1) * cols + ty]
		 + global_conv_kernel3x3[0][2] * input[(tx - 1) * cols + ty + 1]
		 + global_conv_kernel3x3[1][0] * input[tx * cols + ty - 1]
		 + global_conv_kernel3x3[1][1] * input[tx * cols + ty]
		 + global_conv_kernel3x3[1][2] * input[tx * cols + ty + 1]
		 + global_conv_kernel3x3[2][0] * input[(tx + 1) * cols + ty - 1]
		 + global_conv_kernel3x3[2][1] * input[(tx + 1) * cols + ty]
		 + global_conv_kernel3x3[2][2] * input[(tx + 1) * cols + ty + 1]) >> 4;
	}
}

__global__ void k_1D_gf_3x3_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	const int ty = blockIdx.x * blockDim.x + threadIdx.x;
	const int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel3x3[3][3] = { {1, 2, 1},{2, 4, 2},{1, 2, 1} };

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		output[tx * cols + ty] = (conv_kernel3x3[0][0] * input[(tx - 1) * cols + ty - 1]
		 + conv_kernel3x3[0][1] * input[(tx - 1) * cols + ty]
		 + conv_kernel3x3[0][2] * input[(tx - 1) * cols + ty + 1]
		 + conv_kernel3x3[1][0] * input[tx * cols + ty - 1]
		 + conv_kernel3x3[1][1] * input[tx * cols + ty]
		 + conv_kernel3x3[1][2] * input[tx * cols + ty + 1]
		 + conv_kernel3x3[2][0] * input[(tx + 1) * cols + ty - 1]
		 + conv_kernel3x3[2][1] * input[(tx + 1) * cols + ty]
		 + conv_kernel3x3[2][2] * input[(tx + 1) * cols + ty + 1]) >> 4;
	}
}

__global__ void k_1D_gf_3x3_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		output[tx * cols + ty] = (const_conv_kernel3x3[0][0] * input[(tx - 1) * cols + ty - 1]
		+ const_conv_kernel3x3[0][1] * input[(tx - 1) * cols + ty]
		+ const_conv_kernel3x3[0][2] * input[(tx - 1) * cols + ty + 1]
		+ const_conv_kernel3x3[1][0] * input[tx * cols + ty - 1]
		+ const_conv_kernel3x3[1][1] * input[tx * cols + ty]
		+ const_conv_kernel3x3[1][2] * input[tx * cols + ty + 1]
		+ const_conv_kernel3x3[2][0] * input[(tx + 1) * cols + ty - 1]
		+ const_conv_kernel3x3[2][1] * input[(tx + 1) * cols + ty]
		+ const_conv_kernel3x3[2][2] * input[(tx + 1) * cols + ty + 1]) >> 4;
	}
}

__global__ void k_1D_gf_3x3_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][34];

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int cy = threadIdx.x + 1;
	unsigned int cx = threadIdx.y + 1;

    if (tx < rows && ty < cols) {
        cache[cx][cy] = input[tx * cols + ty];
    }

    if (cx == 1 && tx > 0) {
        cache[0][cy] = input[(tx - 1) * cols + ty];
    }
    if (cx == 32 && tx < rows - 1) {
        cache[33][cy] = input[(tx + 1) * cols + ty];
    }
    if (cy == 1 && ty > 0) {
        cache[cx][0] = input[tx * cols + ty - 1];
    }
    if (cy == 32 && ty < cols - 1) {
        cache[cx][33] = input[tx * cols + ty + 1];
    }
	__syncthreads();

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		output[tx * cols + ty] = (global_conv_kernel3x3[0][0] * cache[cx - 1][cy - 1]
		+ global_conv_kernel3x3[0][1] * cache[cx - 1][cy]
		+ global_conv_kernel3x3[0][2] * cache[cx - 1][cy + 1]
		+ global_conv_kernel3x3[1][0] * cache[cx][cy - 1]
		+ global_conv_kernel3x3[1][1] * cache[cx][cy]
		+ global_conv_kernel3x3[1][2] * cache[cx][cy + 1]
		+ global_conv_kernel3x3[2][0] * cache[cx + 1][cy - 1]
		+ global_conv_kernel3x3[2][1] * cache[cx + 1][cy]
		+ global_conv_kernel3x3[2][2] * cache[cx + 1][cy + 1]) >> 4;
	}
}

__global__ void k_1D_gf_3x3_load_balance16_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance12_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;


		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance8_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance4_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
			+ global_conv_kernel3x3[0][1] * frame[0][1]
			+ global_conv_kernel3x3[0][2] * frame[0][2]
			+ global_conv_kernel3x3[1][0] * frame[1][0]
			+ global_conv_kernel3x3[1][1] * frame[1][1]
			+ global_conv_kernel3x3[1][2] * frame[1][2]
			+ global_conv_kernel3x3[2][0] * frame[2][0]
			+ global_conv_kernel3x3[2][1] * frame[2][1]
			+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance2_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance16_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance12_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance8_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance4_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance2_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance16_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
			+ const_conv_kernel3x3[0][1] * frame[0][1]
			+ const_conv_kernel3x3[0][2] * frame[0][2]
			+ const_conv_kernel3x3[1][0] * frame[1][0]
			+ const_conv_kernel3x3[1][1] * frame[1][1]
			+ const_conv_kernel3x3[1][2] * frame[1][2]
			+ const_conv_kernel3x3[2][0] * frame[2][0]
			+ const_conv_kernel3x3[2][1] * frame[2][1]
			+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;


		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
					+ const_conv_kernel3x3[0][1] * frame[0][1]
					+ const_conv_kernel3x3[0][2] * frame[0][2]
					+ const_conv_kernel3x3[1][0] * frame[1][0]
					+ const_conv_kernel3x3[1][1] * frame[1][1]
					+ const_conv_kernel3x3[1][2] * frame[1][2]
					+ const_conv_kernel3x3[2][0] * frame[2][0]
					+ const_conv_kernel3x3[2][1] * frame[2][1]
					+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance12_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];


		output[(tx * cols + ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
			+ const_conv_kernel3x3[0][1] * frame[0][1]
			+ const_conv_kernel3x3[0][2] * frame[0][2]
			+ const_conv_kernel3x3[1][0] * frame[1][0]
			+ const_conv_kernel3x3[1][1] * frame[1][1]
			+ const_conv_kernel3x3[1][2] * frame[1][2]
			+ const_conv_kernel3x3[2][0] * frame[2][0]
			+ const_conv_kernel3x3[2][1] * frame[2][1]
			+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;


		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
					+ const_conv_kernel3x3[0][1] * frame[0][1]
					+ const_conv_kernel3x3[0][2] * frame[0][2]
					+ const_conv_kernel3x3[1][0] * frame[1][0]
					+ const_conv_kernel3x3[1][1] * frame[1][1]
					+ const_conv_kernel3x3[1][2] * frame[1][2]
					+ const_conv_kernel3x3[2][0] * frame[2][0]
					+ const_conv_kernel3x3[2][1] * frame[2][1]
					+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance8_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
			+ const_conv_kernel3x3[0][1] * frame[0][1]
			+ const_conv_kernel3x3[0][2] * frame[0][2]
			+ const_conv_kernel3x3[1][0] * frame[1][0]
			+ const_conv_kernel3x3[1][1] * frame[1][1]
			+ const_conv_kernel3x3[1][2] * frame[1][2]
			+ const_conv_kernel3x3[2][0] * frame[2][0]
			+ const_conv_kernel3x3[2][1] * frame[2][1]
			+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
					+ const_conv_kernel3x3[0][1] * frame[0][1]
					+ const_conv_kernel3x3[0][2] * frame[0][2]
					+ const_conv_kernel3x3[1][0] * frame[1][0]
					+ const_conv_kernel3x3[1][1] * frame[1][1]
					+ const_conv_kernel3x3[1][2] * frame[1][2]
					+ const_conv_kernel3x3[2][0] * frame[2][0]
					+ const_conv_kernel3x3[2][1] * frame[2][1]
					+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance4_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
			+ const_conv_kernel3x3[0][1] * frame[0][1]
			+ const_conv_kernel3x3[0][2] * frame[0][2]
			+ const_conv_kernel3x3[1][0] * frame[1][0]
			+ const_conv_kernel3x3[1][1] * frame[1][1]
			+ const_conv_kernel3x3[1][2] * frame[1][2]
			+ const_conv_kernel3x3[2][0] * frame[2][0]
			+ const_conv_kernel3x3[2][1] * frame[2][1]
			+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
					+ const_conv_kernel3x3[0][1] * frame[0][1]
					+ const_conv_kernel3x3[0][2] * frame[0][2]
					+ const_conv_kernel3x3[1][0] * frame[1][0]
					+ const_conv_kernel3x3[1][1] * frame[1][1]
					+ const_conv_kernel3x3[1][2] * frame[1][2]
					+ const_conv_kernel3x3[2][0] * frame[2][0]
					+ const_conv_kernel3x3[2][1] * frame[2][1]
					+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance2_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		output[(tx * cols + ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
			+ const_conv_kernel3x3[0][1] * frame[0][1]
			+ const_conv_kernel3x3[0][2] * frame[0][2]
			+ const_conv_kernel3x3[1][0] * frame[1][0]
			+ const_conv_kernel3x3[1][1] * frame[1][1]
			+ const_conv_kernel3x3[1][2] * frame[1][2]
			+ const_conv_kernel3x3[2][0] * frame[2][0]
			+ const_conv_kernel3x3[2][1] * frame[2][1]
			+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				output[(tx * cols + _ty)] = (const_conv_kernel3x3[0][0] * frame[0][0]
					+ const_conv_kernel3x3[0][1] * frame[0][1]
					+ const_conv_kernel3x3[0][2] * frame[0][2]
					+ const_conv_kernel3x3[1][0] * frame[1][0]
					+ const_conv_kernel3x3[1][1] * frame[1][1]
					+ const_conv_kernel3x3[1][2] * frame[1][2]
					+ const_conv_kernel3x3[2][0] * frame[2][0]
					+ const_conv_kernel3x3[2][1] * frame[2][1]
					+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance16_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][514];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 16 + 1;
	int cx = threadIdx.y + 1;

	if ((tx > 0 && tx < rows - 1) && (ty < cols - 1)) {
		cache[cx][cy] = input[tx * cols + ty];
		cache[cx][cy + 1] = input[tx * cols + ty + 1];
		cache[cx][cy + 2] = input[tx * cols + ty + 2];
		cache[cx][cy + 3] = input[tx * cols + ty + 3];
		cache[cx][cy + 4] = input[tx * cols + ty + 4];
		cache[cx][cy + 5] = input[tx * cols + ty + 5];
		cache[cx][cy + 6] = input[tx * cols + ty + 6];
		cache[cx][cy + 7] = input[tx * cols + ty + 7];
		cache[cx][cy + 8] = input[tx * cols + ty + 8];
		cache[cx][cy + 9] = input[tx * cols + ty + 9];
		cache[cx][cy + 10] = input[tx * cols + ty + 10];
		cache[cx][cy + 11] = input[tx * cols + ty + 11];
		cache[cx][cy + 12] = input[tx * cols + ty + 12];
		cache[cx][cy + 13] = input[tx * cols + ty + 13];
		cache[cx][cy + 14] = input[tx * cols + ty + 14];
		cache[cx][cy + 15] = input[tx * cols + ty + 15];
		if (cx == 1) { /*top row*/
			cache[0][cy] = input[((tx - 1) * cols + ty)];
			cache[0][cy + 1] = input[((tx - 1) * cols + ty + 1)];
			cache[0][cy + 2] = input[((tx - 1) * cols + ty + 2)];
			cache[0][cy + 3] = input[((tx - 1) * cols + ty + 3)];
			cache[0][cy + 4] = input[((tx - 1) * cols + ty + 4)];
			cache[0][cy + 5] = input[((tx - 1) * cols + ty + 5)];
			cache[0][cy + 6] = input[((tx - 1) * cols + ty + 6)];
			cache[0][cy + 7] = input[((tx - 1) * cols + ty + 7)];
			cache[0][cy + 8] = input[((tx - 1) * cols + ty + 8)];
			cache[0][cy + 9] = input[((tx - 1) * cols + ty + 9)];
			cache[0][cy + 10] = input[((tx - 1) * cols + ty + 10)];
			cache[0][cy + 11] = input[((tx - 1) * cols + ty + 11)];
			cache[0][cy + 12] = input[((tx - 1) * cols + ty + 12)];
			cache[0][cy + 13] = input[((tx - 1) * cols + ty + 13)];
			cache[0][cy + 14] = input[((tx - 1) * cols + ty + 14)];
			cache[0][cy + 15] = input[((tx - 1) * cols + ty + 15)];
		}
		if (cx == 32) { /*bottom row*/
			cache[33][cy] = input[((tx + 1) * cols + ty)];
			cache[33][cy + 1] = input[((tx + 1) * cols + ty + 1)];
			cache[33][cy + 2] = input[((tx + 1) * cols + ty + 2)];
			cache[33][cy + 3] = input[((tx + 1) * cols + ty + 3)];
			cache[33][cy + 4] = input[((tx + 1) * cols + ty + 4)];
			cache[33][cy + 5] = input[((tx + 1) * cols + ty + 5)];
			cache[33][cy + 6] = input[((tx + 1) * cols + ty + 6)];
			cache[33][cy + 7] = input[((tx + 1) * cols + ty + 7)];
			cache[33][cy + 8] = input[((tx + 1) * cols + ty + 8)];
			cache[33][cy + 9] = input[((tx + 1) * cols + ty + 9)];
			cache[33][cy + 10] = input[((tx + 1) * cols + ty + 10)];
			cache[33][cy + 11] = input[((tx + 1) * cols + ty + 11)];
			cache[33][cy + 12] = input[((tx + 1) * cols + ty + 12)];
			cache[33][cy + 13] = input[((tx + 1) * cols + ty + 13)];
			cache[33][cy + 14] = input[((tx + 1) * cols + ty + 14)];
			cache[33][cy + 15] = input[((tx + 1) * cols + ty + 15)];
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 497) {/*right column*/
			cache[cx][513] = input[((tx)*cols + ty + 16)];
		}

		__syncthreads();

		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		output[tx * cols + ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				output[tx * cols + _ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance12_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][386];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 12 + 1;
	int cx = threadIdx.y + 1;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		cache[cx][cy] = input[tx * cols + ty];
		cache[cx][cy + 1] = input[tx * cols + ty + 1];
		cache[cx][cy + 2] = input[tx * cols + ty + 2];
		cache[cx][cy + 3] = input[tx * cols + ty + 3];
		cache[cx][cy + 4] = input[tx * cols + ty + 4];
		cache[cx][cy + 5] = input[tx * cols + ty + 5];
		cache[cx][cy + 6] = input[tx * cols + ty + 6];
		cache[cx][cy + 7] = input[tx * cols + ty + 7];
		cache[cx][cy + 8] = input[tx * cols + ty + 8];
		cache[cx][cy + 9] = input[tx * cols + ty + 9];
		cache[cx][cy + 10] = input[tx * cols + ty + 10];
		cache[cx][cy + 11] = input[tx * cols + ty + 11];
		if (cx == 1) {
			cache[0][cy] = input[((tx - 1) * cols + ty)];
			cache[0][cy + 1] = input[((tx - 1) * cols + ty + 1)];
			cache[0][cy + 2] = input[((tx - 1) * cols + ty + 2)];
			cache[0][cy + 3] = input[((tx - 1) * cols + ty + 3)];
			cache[0][cy + 4] = input[((tx - 1) * cols + ty + 4)];
			cache[0][cy + 5] = input[((tx - 1) * cols + ty + 5)];
			cache[0][cy + 6] = input[((tx - 1) * cols + ty + 6)];
			cache[0][cy + 7] = input[((tx - 1) * cols + ty + 7)];
			cache[0][cy + 8] = input[((tx - 1) * cols + ty + 8)];
			cache[0][cy + 9] = input[((tx - 1) * cols + ty + 9)];
			cache[0][cy + 10] = input[((tx - 1) * cols + ty + 10)];
			cache[0][cy + 11] = input[((tx - 1) * cols + ty + 11)];
		}
		if (cx == 32) { /*bottom row*/
			cache[33][cy] = input[((tx + 1) * cols + ty)];
			cache[33][cy + 1] = input[((tx + 1) * cols + ty + 1)];
			cache[33][cy + 2] = input[((tx + 1) * cols + ty + 2)];
			cache[33][cy + 3] = input[((tx + 1) * cols + ty + 3)];
			cache[33][cy + 4] = input[((tx + 1) * cols + ty + 4)];
			cache[33][cy + 5] = input[((tx + 1) * cols + ty + 5)];
			cache[33][cy + 6] = input[((tx + 1) * cols + ty + 6)];
			cache[33][cy + 7] = input[((tx + 1) * cols + ty + 7)];
			cache[33][cy + 8] = input[((tx + 1) * cols + ty + 8)];
			cache[33][cy + 9] = input[((tx + 1) * cols + ty + 9)];
			cache[33][cy + 10] = input[((tx + 1) * cols + ty + 10)];
			cache[33][cy + 11] = input[((tx + 1) * cols + ty + 11)];
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 373) {/*right column*/
			cache[cx][385] = input[((tx)*cols + ty + 12)];
		}

		__syncthreads();

		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		output[tx * cols + ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				output[tx * cols + _ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance8_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][260];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 8 + 1;
	int cx = threadIdx.y + 1;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		cache[cx][cy] = input[tx * cols + ty];
		cache[cx][cy + 1] = input[tx * cols + ty + 1];
		cache[cx][cy + 2] = input[tx * cols + ty + 2];
		cache[cx][cy + 3] = input[tx * cols + ty + 3];
		cache[cx][cy + 4] = input[tx * cols + ty + 4];
		cache[cx][cy + 5] = input[tx * cols + ty + 5];
		cache[cx][cy + 6] = input[tx * cols + ty + 6];
		cache[cx][cy + 7] = input[tx * cols + ty + 7];
		if (cx == 1) { 
			cache[0][cy] = input[((tx - 1) * cols + ty)];
			cache[0][cy + 1] = input[((tx - 1) * cols + ty + 1)];
			cache[0][cy + 2] = input[((tx - 1) * cols + ty + 2)];
			cache[0][cy + 3] = input[((tx - 1) * cols + ty + 3)];
			cache[0][cy + 4] = input[((tx - 1) * cols + ty + 4)];
			cache[0][cy + 5] = input[((tx - 1) * cols + ty + 5)];
			cache[0][cy + 6] = input[((tx - 1) * cols + ty + 6)];
			cache[0][cy + 7] = input[((tx - 1) * cols + ty + 7)];
		}
		if (cx == 32) { /*bottom row*/
			cache[33][cy] = input[((tx + 1) * cols + ty)];
			cache[33][cy + 1] = input[((tx + 1) * cols + ty + 1)];
			cache[33][cy + 2] = input[((tx + 1) * cols + ty + 2)];
			cache[33][cy + 3] = input[((tx + 1) * cols + ty + 3)];
			cache[33][cy + 4] = input[((tx + 1) * cols + ty + 4)];
			cache[33][cy + 5] = input[((tx + 1) * cols + ty + 5)];
			cache[33][cy + 6] = input[((tx + 1) * cols + ty + 6)];
			cache[33][cy + 7] = input[((tx + 1) * cols + ty + 7)];
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 249) {/*right column*/
			cache[cx][257] = input[((tx)*cols + ty + 8)];
		}
		__syncthreads();

		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		output[tx * cols + ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;


		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				output[tx * cols + _ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance4_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][130];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 4 + 1;
	int cx = threadIdx.y + 1;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
	cache[cx][cy] = input[tx * cols + ty];
	cache[cx][cy + 1] = input[tx * cols + ty + 1];
	cache[cx][cy + 2] = input[tx * cols + ty + 2];
	cache[cx][cy + 3] = input[tx * cols + ty + 3];
		
		if (cx == 1) {
			cache[0][cy] = input[((tx - 1) * cols + ty)];
			cache[0][cy + 1] = input[((tx - 1) * cols + ty + 1)];
			cache[0][cy + 2] = input[((tx - 1) * cols + ty + 2)];
			cache[0][cy + 3] = input[((tx - 1) * cols + ty + 3)];
		}
		if (cx == 32) { /*bottom row*/
			cache[33][cy] = input[((tx + 1) * cols + ty)];
			cache[33][cy + 1] = input[((tx + 1) * cols + ty + 1)];
			cache[33][cy + 2] = input[((tx + 1) * cols + ty + 2)];
			cache[33][cy + 3] = input[((tx + 1) * cols + ty + 3)];
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 125) {/*right column*/
			cache[cx][129] = input[((tx)*cols + ty + 4)];
		}
		__syncthreads();

		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		output[tx * cols + ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				output[tx * cols + _ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_load_balance2_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][66];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 2 + 1;
	int cx = threadIdx.y + 1;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		cache[cx][cy] = input[tx * cols + ty];
		cache[cx][cy + 1] = input[tx * cols + ty + 1];

		if (cx == 1) { /*top row*/
			cache[0][cy] = input[((tx - 1) * cols + ty)];
			cache[0][cy + 1] = input[((tx - 1) * cols + ty + 1)];
		}
		if (cx == 32) { /*bottom row*/
			cache[33][cy] = input[((tx + 1) * cols + ty)];
			cache[33][cy + 1] = input[((tx + 1) * cols + ty + 1)];
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 63) {/*right column*/
			cache[cx][65] = input[((tx)*cols + ty + 2)];
		}
		__syncthreads();

		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		output[tx * cols + ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if ( _ty < cols - 1) {
				output[tx * cols + _ty] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
	}
}

__global__ void k_1D_gf_3x3_vectorized16_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[16] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty  < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
			+ global_conv_kernel3x3[0][1] * frame[0][1]
			+ global_conv_kernel3x3[0][2] * frame[0][2]
			+ global_conv_kernel3x3[1][0] * frame[1][0]
			+ global_conv_kernel3x3[1][1] * frame[1][1]
			+ global_conv_kernel3x3[1][2] * frame[1][2]
			+ global_conv_kernel3x3[2][0] * frame[2][0]
			+ global_conv_kernel3x3[2][1] * frame[2][1]
			+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8], vals[9], vals[10], vals[11]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 12)])[0] = make_uchar4(vals[12], vals[13], vals[14], vals[15]);
	}
}

__global__ void k_1D_gf_3x3_vectorized12_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[12] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
			+ global_conv_kernel3x3[0][1] * frame[0][1]
			+ global_conv_kernel3x3[0][2] * frame[0][2]
			+ global_conv_kernel3x3[1][0] * frame[1][0]
			+ global_conv_kernel3x3[1][1] * frame[1][1]
			+ global_conv_kernel3x3[1][2] * frame[1][2]
			+ global_conv_kernel3x3[2][0] * frame[2][0]
			+ global_conv_kernel3x3[2][1] * frame[2][1]
			+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8], vals[9], vals[10], vals[11]);
	}
}
__global__ void k_1D_gf_3x3_vectorized8_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[8] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
			+ global_conv_kernel3x3[0][1] * frame[0][1]
			+ global_conv_kernel3x3[0][2] * frame[0][2]
			+ global_conv_kernel3x3[1][0] * frame[1][0]
			+ global_conv_kernel3x3[1][1] * frame[1][1]
			+ global_conv_kernel3x3[1][2] * frame[1][2]
			+ global_conv_kernel3x3[2][0] * frame[2][0]
			+ global_conv_kernel3x3[2][1] * frame[2][1]
			+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
	}
}

__global__ void k_1D_gf_3x3_vectorized4_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[4] = { 0 };

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
			+ global_conv_kernel3x3[0][1] * frame[0][1]
			+ global_conv_kernel3x3[0][2] * frame[0][2]
			+ global_conv_kernel3x3[1][0] * frame[1][0]
			+ global_conv_kernel3x3[1][1] * frame[1][1]
			+ global_conv_kernel3x3[1][2] * frame[1][2]
			+ global_conv_kernel3x3[2][0] * frame[2][0]
			+ global_conv_kernel3x3[2][1] * frame[2][1]
			+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4; 

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
	}
}

__global__ void k_1D_gf_3x3_vectorized2_global(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[2] = { 0 };

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
			+ global_conv_kernel3x3[0][1] * frame[0][1]
			+ global_conv_kernel3x3[0][2] * frame[0][2]
			+ global_conv_kernel3x3[1][0] * frame[1][0]
			+ global_conv_kernel3x3[1][1] * frame[1][1]
			+ global_conv_kernel3x3[1][2] * frame[1][2]
			+ global_conv_kernel3x3[2][0] * frame[2][0]
			+ global_conv_kernel3x3[2][1] * frame[2][1]
			+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar2*>(&output[(tx * cols + ty)])[0] = make_uchar2(vals[0], vals[1]);
	}
}
__global__ void k_1D_gf_3x3_vectorized16_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };
	int vals[16] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8], vals[9], vals[10], vals[11]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 12)])[0] = make_uchar4(vals[12], vals[13], vals[14], vals[15]);
	}
}

__global__ void k_1D_gf_3x3_vectorized12_local(unsigned char* input, unsigned char* output, int rows, int cols){
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };
	int vals[12] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8], vals[9], vals[10], vals[11]);
	}
}
__global__ void k_1D_gf_3x3_vectorized8_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };
	int vals[8] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
	}
}

__global__ void k_1D_gf_3x3_vectorized4_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };
	int vals[4] = { 0 };

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
	}
}

__global__ void k_1D_gf_3x3_vectorized2_local(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char conv_kernel[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };
	int vals[2] = { 0 };

	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (conv_kernel[0][0] * frame[0][0]
			+ conv_kernel[0][1] * frame[0][1]
			+ conv_kernel[0][2] * frame[0][2]
			+ conv_kernel[1][0] * frame[1][0]
			+ conv_kernel[1][1] * frame[1][1]
			+ conv_kernel[1][2] * frame[1][2]
			+ conv_kernel[2][0] * frame[2][0]
			+ conv_kernel[2][1] * frame[2][1]
			+ conv_kernel[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (conv_kernel[0][0] * frame[0][0]
					+ conv_kernel[0][1] * frame[0][1]
					+ conv_kernel[0][2] * frame[0][2]
					+ conv_kernel[1][0] * frame[1][0]
					+ conv_kernel[1][1] * frame[1][1]
					+ conv_kernel[1][2] * frame[1][2]
					+ conv_kernel[2][0] * frame[2][0]
					+ conv_kernel[2][1] * frame[2][1]
					+ conv_kernel[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar2*>(&output[(tx * cols + ty)])[0] = make_uchar2(vals[0], vals[1]);
	}
}

__global__ void k_1D_gf_3x3_vectorized16_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[16] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = (const_conv_kernel3x3[0][0] * frame[0][0]
			+ const_conv_kernel3x3[0][1] * frame[0][1]
			+ const_conv_kernel3x3[0][2] * frame[0][2]
			+ const_conv_kernel3x3[1][0] * frame[1][0]
			+ const_conv_kernel3x3[1][1] * frame[1][1]
			+ const_conv_kernel3x3[1][2] * frame[1][2]
			+ const_conv_kernel3x3[2][0] * frame[2][0]
			+ const_conv_kernel3x3[2][1] * frame[2][1]
			+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = (const_conv_kernel3x3[0][0] * frame[0][0]
					+ const_conv_kernel3x3[0][1] * frame[0][1]
					+ const_conv_kernel3x3[0][2] * frame[0][2]
					+ const_conv_kernel3x3[1][0] * frame[1][0]
					+ const_conv_kernel3x3[1][1] * frame[1][1]
					+ const_conv_kernel3x3[1][2] * frame[1][2]
					+ const_conv_kernel3x3[2][0] * frame[2][0]
					+ const_conv_kernel3x3[2][1] * frame[2][1]
					+ const_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8], vals[9], vals[10], vals[11]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 12)])[0] = make_uchar4(vals[12], vals[13], vals[14], vals[15]);
	}
}

__global__ void k_1D_gf_3x3_vectorized12_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[12] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = const_conv_kernel3x3[0][0] * frame[0][0]
		+ const_conv_kernel3x3[0][1] * frame[0][1]
		+ const_conv_kernel3x3[0][2] * frame[0][2]
		+ const_conv_kernel3x3[1][0] * frame[1][0]
		+ const_conv_kernel3x3[1][1] * frame[1][1]
		+ const_conv_kernel3x3[1][2] * frame[1][2]
		+ const_conv_kernel3x3[2][0] * frame[2][0]
		+ const_conv_kernel3x3[2][1] * frame[2][1]
		+ const_conv_kernel3x3[2][2] * frame[2][2];

		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = const_conv_kernel3x3[0][0] * frame[0][0]
				+ const_conv_kernel3x3[0][1] * frame[0][1]
				+ const_conv_kernel3x3[0][2] * frame[0][2]
				+ const_conv_kernel3x3[1][0] * frame[1][0]
				+ const_conv_kernel3x3[1][1] * frame[1][1]
				+ const_conv_kernel3x3[1][2] * frame[1][2]
				+ const_conv_kernel3x3[2][0] * frame[2][0]
				+ const_conv_kernel3x3[2][1] * frame[2][1]
				+ const_conv_kernel3x3[2][2] * frame[2][2];
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0] >> 4, vals[1] >> 4, vals[2] >> 4, vals[3] >> 4);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4] >> 4, vals[5] >> 4, vals[6] >> 4, vals[7] >> 4);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8] >> 4, vals[9] >> 4, vals[10] >> 4, vals[11] >> 4);
	}
}
__global__ void k_1D_gf_3x3_vectorized8_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[8] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = const_conv_kernel3x3[0][0] * frame[0][0]
		+ const_conv_kernel3x3[0][1] * frame[0][1]
		+ const_conv_kernel3x3[0][2] * frame[0][2]
		+ const_conv_kernel3x3[1][0] * frame[1][0]
		+ const_conv_kernel3x3[1][1] * frame[1][1]
		+ const_conv_kernel3x3[1][2] * frame[1][2]
		+ const_conv_kernel3x3[2][0] * frame[2][0]
		+ const_conv_kernel3x3[2][1] * frame[2][1]
		+ const_conv_kernel3x3[2][2] * frame[2][2];

		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = const_conv_kernel3x3[0][0] * frame[0][0]
				+ const_conv_kernel3x3[0][1] * frame[0][1]
				+ const_conv_kernel3x3[0][2] * frame[0][2]
				+ const_conv_kernel3x3[1][0] * frame[1][0]
				+ const_conv_kernel3x3[1][1] * frame[1][1]
				+ const_conv_kernel3x3[1][2] * frame[1][2]
				+ const_conv_kernel3x3[2][0] * frame[2][0]
				+ const_conv_kernel3x3[2][1] * frame[2][1]
				+ const_conv_kernel3x3[2][2] * frame[2][2];
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0] >> 4, vals[1] >> 4, vals[2] >> 4, vals[3] >> 4);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4] >> 4, vals[5] >> 4, vals[6] >> 4, vals[7] >> 4);
	}
}

__global__ void k_1D_gf_3x3_vectorized4_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[4] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = const_conv_kernel3x3[0][0] * frame[0][0]
		+ const_conv_kernel3x3[0][1] * frame[0][1]
		+ const_conv_kernel3x3[0][2] * frame[0][2]
		+ const_conv_kernel3x3[1][0] * frame[1][0]
		+ const_conv_kernel3x3[1][1] * frame[1][1]
		+ const_conv_kernel3x3[1][2] * frame[1][2]
		+ const_conv_kernel3x3[2][0] * frame[2][0]
		+ const_conv_kernel3x3[2][1] * frame[2][1]
		+ const_conv_kernel3x3[2][2] * frame[2][2];

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = const_conv_kernel3x3[0][0] * frame[0][0]
				+ const_conv_kernel3x3[0][1] * frame[0][1]
				+ const_conv_kernel3x3[0][2] * frame[0][2]
				+ const_conv_kernel3x3[1][0] * frame[1][0]
				+ const_conv_kernel3x3[1][1] * frame[1][1]
				+ const_conv_kernel3x3[1][2] * frame[1][2]
				+ const_conv_kernel3x3[2][0] * frame[2][0]
				+ const_conv_kernel3x3[2][1] * frame[2][1]
				+ const_conv_kernel3x3[2][2] * frame[2][2];
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0] >> 4, vals[1] >> 4, vals[2] >> 4, vals[3] >> 4);
	}
}

__global__ void k_1D_gf_3x3_vectorized2_constant(unsigned char* input, unsigned char* output, int rows, int cols)
{
	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int vals[2] = { 0 };
	unsigned char frame[3][3];

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		frame[0][0] = input[(tx - 1) * cols + ty - 1];
		frame[0][1] = input[(tx - 1) * cols + ty];
		frame[0][2] = input[(tx - 1) * cols + ty + 1];
		frame[1][0] = input[tx * cols + ty - 1];
		frame[1][1] = input[tx * cols + ty];
		frame[1][2] = input[tx * cols + ty + 1];
		frame[2][0] = input[(tx + 1) * cols + ty - 1];
		frame[2][1] = input[(tx + 1) * cols + ty];
		frame[2][2] = input[(tx + 1) * cols + ty + 1];

		vals[0] = const_conv_kernel3x3[0][0] * frame[0][0]
		+ const_conv_kernel3x3[0][1] * frame[0][1]
		+ const_conv_kernel3x3[0][2] * frame[0][2]
		+ const_conv_kernel3x3[1][0] * frame[1][0]
		+ const_conv_kernel3x3[1][1] * frame[1][1]
		+ const_conv_kernel3x3[1][2] * frame[1][2]
		+ const_conv_kernel3x3[2][0] * frame[2][0]
		+ const_conv_kernel3x3[2][1] * frame[2][1]
		+ const_conv_kernel3x3[2][2] * frame[2][2];
		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			shift_left(frame);
			if ((tx > 0 && tx < rows - 1) && (_ty > 0 && _ty < cols - 1)) {
				frame[0][2] = input[(tx - 1) * cols + _ty + 1];
				frame[1][2] = input[tx * cols + _ty + 1];
				frame[2][2] = input[(tx + 1) * cols + _ty + 1];

				vals[i] = const_conv_kernel3x3[0][0] * frame[0][0]
				+ const_conv_kernel3x3[0][1] * frame[0][1]
				+ const_conv_kernel3x3[0][2] * frame[0][2]
				+ const_conv_kernel3x3[1][0] * frame[1][0]
				+ const_conv_kernel3x3[1][1] * frame[1][1]
				+ const_conv_kernel3x3[1][2] * frame[1][2]
				+ const_conv_kernel3x3[2][0] * frame[2][0]
				+ const_conv_kernel3x3[2][1] * frame[2][1]
				+ const_conv_kernel3x3[2][2] * frame[2][2];
			}
		}
		reinterpret_cast<uchar2*>(&output[(tx * cols + ty)])[0] = make_uchar2(vals[0] >> 4, vals[1] >> 4);
	}
}

__global__ void k_1D_gf_3x3_vectorized16_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][514];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 16 + 1;
	int cx = threadIdx.y + 1;

	uchar4 u4;

	if ((tx > 0 && tx < rows - 1) && (ty < cols - 1)) {
		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty])[0];
		cache[cx][cy] = u4.x;
		cache[cx][cy + 1] = u4.y;
		cache[cx][cy + 2] = u4.z;
		cache[cx][cy + 3] = u4.w;

		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty + 4])[0];
		cache[cx][cy + 4] = u4.x;
		cache[cx][cy + 5] = u4.y;
		cache[cx][cy + 6] = u4.z;
		cache[cx][cy + 7] = u4.w;

		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty + 8])[0];
		cache[cx][cy + 8] = u4.x;
		cache[cx][cy + 9] = u4.y;
		cache[cx][cy + 10] = u4.z;
		cache[cx][cy + 11] = u4.w;

		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty + 12])[0];
		cache[cx][cy + 12] = u4.x;
		cache[cx][cy + 13] = u4.y;
		cache[cx][cy + 14] = u4.z;
		cache[cx][cy + 15] = u4.w;

		if (cx == 1) { /*top row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty])[0];
			cache[0][cy] = u4.x;
			cache[0][cy + 1] = u4.y;
			cache[0][cy + 2] = u4.z;
			cache[0][cy + 3] = u4.w;

			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty + 4])[0];
			cache[0][cy + 4] = u4.x;
			cache[0][cy + 5] = u4.y;
			cache[0][cy + 6] = u4.z;
			cache[0][cy + 7] = u4.w;

			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty + 8])[0];
			cache[0][cy + 8] = u4.x;
			cache[0][cy + 9] = u4.y;
			cache[0][cy + 10] = u4.z;
			cache[0][cy + 11] = u4.w;

			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty + 12])[0];
			cache[0][cy + 12] = u4.x;
			cache[0][cy + 13] = u4.y;
			cache[0][cy + 14] = u4.z;
			cache[0][cy + 15] = u4.w;
		}
		if (cx == 32) { /*bottom row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty])[0];
			cache[33][cy] = u4.x;
			cache[33][cy + 1] = u4.y;
			cache[33][cy + 2] = u4.z;
			cache[33][cy + 3] = u4.w;
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty + 4])[0];
			cache[33][cy + 4] = u4.x;
			cache[33][cy + 5] = u4.y;
			cache[33][cy + 6] = u4.z;
			cache[33][cy + 7] = u4.w;
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty + 8])[0];
			cache[33][cy + 8] = u4.x;
			cache[33][cy + 9] = u4.y;
			cache[33][cy + 10] = u4.z;
			cache[33][cy + 11] = u4.w;
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty + 12])[0];
			cache[33][cy + 12] = u4.x;
			cache[33][cy + 13] = u4.y;
			cache[33][cy + 14] = u4.z;
			cache[33][cy + 15] = u4.w;
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 497) {/*right column*/
			cache[cx][513] = input[((tx)*cols + ty + 16)];
		}

		__syncthreads();
		int vals[16] = { 0 };
		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 16; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8], vals[9], vals[10], vals[11]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 12)])[0] = make_uchar4(vals[12], vals[13], vals[14], vals[15]);
	}
}

__global__ void k_1D_gf_3x3_vectorized12_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][386];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 12;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 12 + 1;
	int cx = threadIdx.y + 1;
	
	uchar4 u4;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty])[0];
		cache[cx][cy] = u4.x;
		cache[cx][cy + 1] = u4.y;
		cache[cx][cy + 2] = u4.z;
		cache[cx][cy + 3] = u4.w;

		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty + 4])[0];
		cache[cx][cy + 4] = u4.x;
		cache[cx][cy + 5] = u4.y;
		cache[cx][cy + 6] = u4.z;
		cache[cx][cy + 7] = u4.w;

		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty + 8])[0];
		cache[cx][cy + 8] = u4.x;
		cache[cx][cy + 9] = u4.y;
		cache[cx][cy + 10] = u4.z;
		cache[cx][cy + 11] = u4.w;
		if (cx == 1) { /*top row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1 ) * cols + ty])[0];
			cache[0][cy] = u4.x;
			cache[0][cy + 1] = u4.y;
			cache[0][cy + 2] = u4.z;
			cache[0][cy + 3] = u4.w;

			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty + 4])[0];
			cache[0][cy + 4] = u4.x;
			cache[0][cy + 5] = u4.y;
			cache[0][cy + 6] = u4.z;
			cache[0][cy + 7] = u4.w;

			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty + 8])[0];
			cache[0][cy + 8] = u4.x;
			cache[0][cy + 9] = u4.y;
			cache[0][cy + 10] = u4.z; 
			cache[0][cy + 11] = u4.w; 
		}
		if (cx == 32) { /*bottom row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty])[0];
			cache[33][cy] = u4.x;
			cache[33][cy + 1] = u4.y;
			cache[33][cy + 2] = u4.z;
			cache[33][cy + 3] = u4.w;
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty + 4])[0];
			cache[33][cy + 4] = u4.x;
			cache[33][cy + 5] = u4.y;
			cache[33][cy + 6] = u4.z;
			cache[33][cy + 7] = u4.w;
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty + 8])[0];
			cache[33][cy + 8] = u4.x;
			cache[33][cy + 9] = u4.y;
			cache[33][cy + 10] = u4.z;
			cache[33][cy + 11] = u4.w;
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 373) {/*right column*/
			cache[cx][385] = input[((tx)*cols + ty + 12)];
		}
		__syncthreads();
		int vals[12] = { 0 };
		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2] >> 4);

		for (int i = 1; i < 12; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 8)])[0] = make_uchar4(vals[8], vals[9], vals[10], vals[11]);
	}
}

__global__ void k_1D_gf_3x3_vectorized8_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][260];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 8 + 1;
	int cx = threadIdx.y + 1;

	uchar4 u4;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty])[0];
		cache[cx][cy] = u4.x;
		cache[cx][cy + 1] = u4.y;
		cache[cx][cy + 2] = u4.z;
		cache[cx][cy + 3] = u4.w;

		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty + 4])[0];
		cache[cx][cy + 4] = u4.x;
		cache[cx][cy + 5] = u4.y;
		cache[cx][cy + 6] = u4.z;
		cache[cx][cy + 7] = u4.w;
		if (cx == 1) { /*top row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty])[0];
			cache[0][cy] = u4.x;
			cache[0][cy + 1] = u4.y;
			cache[0][cy + 2] = u4.z;
			cache[0][cy + 3] = u4.w;
			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty + 4])[0];
			cache[0][cy + 4] = u4.x;
			cache[0][cy + 5] = u4.y;
			cache[0][cy + 6] = u4.z;
			cache[0][cy + 7] = u4.w;
		}
		if (cx == 32) { /*bottom row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty])[0];
			cache[33][cy] = u4.x;
			cache[33][cy + 1] = u4.y;
			cache[33][cy + 2] = u4.z;
			cache[33][cy + 3] = u4.w;
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty + 4])[0];
			cache[33][cy + 4] = u4.x;
			cache[33][cy + 5] = u4.y;
			cache[33][cy + 6] = u4.z;
			cache[33][cy + 7] = u4.w;
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[((tx)*cols + ty - 1)];
		}
		if (cy == 249) {/*right column*/
			cache[cx][257] = input[((tx)*cols + ty + 8)];
		}
		__syncthreads();
		int vals[8] = { 0 };
		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 8; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty + 4)])[0] = make_uchar4(vals[4], vals[5], vals[6], vals[7]);
	}
}

__global__ void k_1D_gf_3x3_vectorized4_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][130];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 4 + 1;
	int cx = threadIdx.y + 1;

	uchar4 u4;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		u4 = reinterpret_cast<uchar4*>(&input[tx * cols + ty])[0];
		cache[cx][cy] = u4.x;
		cache[cx][cy + 1] = u4.y;
		cache[cx][cy + 2] = u4.z;
		cache[cx][cy + 3] = u4.w;
		if (cx == 1) { /*top row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx - 1) * cols + ty])[0];
			cache[0][cy] = u4.x;
			cache[0][cy + 1] = u4.y;
			cache[0][cy + 2] = u4.z;
			cache[0][cy + 3] = u4.w;
		}
		if (cx == 32) { /*bottom row*/
			u4 = reinterpret_cast<uchar4*>(&input[(tx + 1) * cols + ty])[0];
			cache[33][cy] = u4.x;
			cache[33][cy + 1] = u4.y;
			cache[33][cy + 2] = u4.z;
			cache[33][cy + 3] = u4.w;
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[(tx * cols + ty - 1)];
		}
		if (cy == 125) {/*right column*/
			cache[cx][129] = input[(tx * cols + ty + 4)];
		}
		__syncthreads();
		int vals[4] = { 0 };
		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 4; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
				+ global_conv_kernel3x3[0][1] * frame[0][1]
				+ global_conv_kernel3x3[0][2] * frame[0][2]
				+ global_conv_kernel3x3[1][0] * frame[1][0]
				+ global_conv_kernel3x3[1][1] * frame[1][1]
				+ global_conv_kernel3x3[1][2] * frame[1][2]
				+ global_conv_kernel3x3[2][0] * frame[2][0]
				+ global_conv_kernel3x3[2][1] * frame[2][1]
				+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar4*>(&output[(tx * cols + ty)])[0] = make_uchar4(vals[0], vals[1], vals[2], vals[3]);
	}
}


__global__ void k_1D_gf_3x3_vectorized2_shared(unsigned char* input, unsigned char* output, int rows, int cols)
{
	__shared__  unsigned char cache[34][66];

	int ty = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int cy = threadIdx.x * 2 + 1;
	int cx = threadIdx.y + 1;

	uchar2 u2;

	if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1)) {
		u2 = reinterpret_cast<uchar2*>(&input[tx * cols + ty])[0];
		
		cache[cx][cy] = u2.x;
		cache[cx][cy + 1] = u2.y;
		if (cx == 1) { /*top row*/
			u2 = reinterpret_cast<uchar2*>(&input[(tx - 1) * cols + ty])[0];
			cache[0][cy] = u2.x;
			cache[0][cy + 1] = u2.y;
		}
		if (cx == 32) { /*bottom row*/
			u2 = reinterpret_cast<uchar2*>(&input[(tx + 1) * cols + ty])[0];
			cache[33][cy] = u2.x;
			cache[33][cy + 1] = u2.y;
		}
		if (cy == 1) {/*left column*/
			cache[cx][0] = input[(tx * cols + ty - 1)];
		}
		if (cy == 63) {/*right column*/
			cache[cx][65] = input[((tx)*cols + ty + 2)];
		}
		__syncthreads();
		int vals[2] = { 0 };
		unsigned char frame[3][3];

		frame[0][0] = cache[cx - 1][cy - 1];
		frame[0][1] = cache[cx - 1][cy];
		frame[0][2] = cache[cx - 1][cy + 1];
		frame[1][0] = cache[cx][cy - 1];
		frame[1][1] = cache[cx][cy];
		frame[1][2] = cache[cx][cy + 1];
		frame[2][0] = cache[cx + 1][cy - 1];
		frame[2][1] = cache[cx + 1][cy];
		frame[2][2] = cache[cx + 1][cy + 1];

		vals[0] = (global_conv_kernel3x3[0][0] * frame[0][0]
		+ global_conv_kernel3x3[0][1] * frame[0][1]
		+ global_conv_kernel3x3[0][2] * frame[0][2]
		+ global_conv_kernel3x3[1][0] * frame[1][0]
		+ global_conv_kernel3x3[1][1] * frame[1][1]
		+ global_conv_kernel3x3[1][2] * frame[1][2]
		+ global_conv_kernel3x3[2][0] * frame[2][0]
		+ global_conv_kernel3x3[2][1] * frame[2][1]
		+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;

		for (int i = 1; i < 2; i++) {
			int _ty = ty + i;
			int _cy = cy + i;
			shift_left(frame);
			frame[0][2] = cache[cx - 1][_cy + 1];
			frame[1][2] = cache[cx][_cy + 1];
			frame[2][2] = cache[cx + 1][_cy + 1];

			if (_ty < cols - 1) {
				vals[i] = (global_conv_kernel3x3[0][0] * frame[0][0]
					+ global_conv_kernel3x3[0][1] * frame[0][1]
					+ global_conv_kernel3x3[0][2] * frame[0][2]
					+ global_conv_kernel3x3[1][0] * frame[1][0]
					+ global_conv_kernel3x3[1][1] * frame[1][1]
					+ global_conv_kernel3x3[1][2] * frame[1][2]
					+ global_conv_kernel3x3[2][0] * frame[2][0]
					+ global_conv_kernel3x3[2][1] * frame[2][1]
					+ global_conv_kernel3x3[2][2] * frame[2][2]) >> 4;
			}
		}
		reinterpret_cast<uchar2*>(&output[(tx * cols + ty)])[0] = make_uchar2(vals[0], vals[1]);
	}
}

void gf_1d_gpu(cv::Mat* input_img, cv::Mat* output_img, GAUSSIAN ver)
{
	unsigned char* d_input = nullptr;
	unsigned char* d_output = nullptr;
	unsigned char* h_input = input_img->data;
	unsigned char* h_output = output_img->data;

	const int cols = (*input_img).cols;
	const int rows = (*input_img).rows;
	const int size = cols * rows * sizeof(unsigned char);

	const unsigned char conv_kernel3x3[3][3] = { {1, 2, 1}, {2, 4, 2}, {1, 2, 1} };

	dim3 block(32,32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	dim3 grid2(((cols / 2) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	dim3 grid4(((cols / 4) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	dim3 grid8(((cols / 8) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	dim3 grid12(((cols / 12) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	dim3 grid16(((cols / 16) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	dim3 grid32(((cols / 32) + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	hipHostRegister(h_output, size, hipHostRegisterPortable);
	hipHostRegister(h_input, size, hipHostRegisterPortable);
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_input, size));
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_output, size));
	CHECK_CUDA_ERROR(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemset((void*)d_output, 0, size));

	switch (ver)
	{
	default:
		break;
	case GAUSSIAN_3x3_global:
		k_1D_gf_3x3_global << <grid, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_local:
		k_1D_gf_3x3_local << <grid, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_constant << <grid, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_shared:
		k_1D_gf_3x3_shared << <grid, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance16_global:
		k_1D_gf_3x3_load_balance16_global << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance12_global:
		k_1D_gf_3x3_load_balance12_global << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance8_global:
		k_1D_gf_3x3_load_balance8_global << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance4_global:
		k_1D_gf_3x3_load_balance4_global << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance2_global:
		k_1D_gf_3x3_load_balance2_global << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized16_global:
		k_1D_gf_3x3_vectorized16_global << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized12_global:
		k_1D_gf_3x3_vectorized12_global << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized8_global:
		k_1D_gf_3x3_vectorized8_global << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized4_global:
		k_1D_gf_3x3_vectorized4_global << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized2_global:
		k_1D_gf_3x3_vectorized2_global << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance16_local:
		k_1D_gf_3x3_load_balance16_local << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance12_local:
		k_1D_gf_3x3_load_balance12_local << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance8_local:
		k_1D_gf_3x3_load_balance8_local << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance4_local:
		k_1D_gf_3x3_load_balance4_local << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance2_local:
		k_1D_gf_3x3_load_balance2_local << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized16_local:
		k_1D_gf_3x3_vectorized16_local << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized12_local:
		k_1D_gf_3x3_vectorized12_local << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized8_local:
		k_1D_gf_3x3_vectorized8_local << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized4_local:
		k_1D_gf_3x3_vectorized4_local << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized2_local:
		k_1D_gf_3x3_vectorized2_local << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance16_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_load_balance16_constant << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance12_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_load_balance12_constant << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance8_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_load_balance8_constant << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance4_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_load_balance4_constant << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance2_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_load_balance2_constant << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized16_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_vectorized16_constant << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized12_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_vectorized12_constant << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized8_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_vectorized8_constant << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized4_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_vectorized4_constant << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized2_constant:
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel3x3), conv_kernel3x3, sizeof(unsigned char) * 3 * 3));
		k_1D_gf_3x3_vectorized2_constant << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance16_shared:
		k_1D_gf_3x3_load_balance16_shared << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance12_shared:
		k_1D_gf_3x3_load_balance12_shared << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance8_shared:
		k_1D_gf_3x3_load_balance8_shared << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance4_shared:
		k_1D_gf_3x3_load_balance4_shared << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_load_balance2_shared:
		k_1D_gf_3x3_load_balance2_shared << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized16_shared:
		k_1D_gf_3x3_vectorized16_shared << <grid16, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized12_shared:
		k_1D_gf_3x3_vectorized12_shared << <grid12, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized8_shared:
		k_1D_gf_3x3_vectorized8_shared << <grid8, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized4_shared:
		k_1D_gf_3x3_vectorized4_shared << <grid4, block >> > (d_input, d_output, rows, cols);
		break;
	case GAUSSIAN_3x3_vectorized2_shared:
		k_1D_gf_3x3_vectorized2_shared << <grid2, block >> > (d_input, d_output, rows, cols);
		break;
	}

	CHECK_CUDA_ERROR(hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost));

	hipHostUnregister(h_input);
	hipHostUnregister(h_output);
	hipFree(d_input);
	hipFree(d_output);
	hipDeviceReset();

}